#include "hip/hip_runtime.h"
#include <type_traits>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/nested/NestedTensorTransformerFunctions.h>
#include <ATen/native/nested/NestedTensorUtils.h>

#ifndef USE_ROCM
#ifndef _WIN32
#include <cutlass/gemm/device/default_gemm_configuration.h>
#include <cutlass/gemm/device/gemm_grouped.h>
#include <cutlass/gemm/kernel/default_gemm_grouped.h>
#endif
#endif

#include <ATen/NestedTensorImpl.h>

#define BLOCK_DIM 256
#define GRID_DIM_Y 16

namespace at {
namespace native {

template <typename T>
__global__ void remove_padding_transform0213_2(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  int input_offset =
      batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i2 = i / sizes_i[1];
    const int i13 = i % sizes_i[1];
    const int i1 = i13 / (sizes_i[1] / input_sizes[1]);
    const int i3 = i13 % (sizes_i[1] / input_sizes[1]);

    output[offset + i] = input
        [input_offset + i1 * input_sizes[2] * input_sizes[3] +
         i2 * input_sizes[3] + i3];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i2 = i / sizes_i[1];
    const int i13 = i % sizes_i[1];
    const int i1 = i13 / (sizes_i[1] / input_sizes[1]);
    const int i3 = i13 % (sizes_i[1] / input_sizes[1]);
    output[offset + i] = input
        [input_offset + i1 * input_sizes[2] * input_sizes[3] +
         i2 * input_sizes[3] + i3];
  }
}

template <typename T>
__global__ void remove_padding_2(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  int input_offset = batch_id * input_sizes[1] * input_sizes[2];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / sizes_i[1];
    const int i1 = i % sizes_i[1];
    const int i0_offset = i0 * input_sizes[2];
    output[offset + i] = input[input_offset + i0_offset + i1];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / sizes_i[1];
    const int i1 = i % sizes_i[1];
    const int i0_offset = i0 * input_sizes[2];
    output[offset + i] = input[input_offset + i0_offset + i1];
  }
}

template <typename T>
__global__ void remove_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1] * sizes_i[2];
  int input_offset =
      batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
}

template <typename T>
void remove_padding_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  dim3 grid;
  grid.x = batch_size;
  grid.y = GRID_DIM_Y;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  if (output_dim == 2) {
    remove_padding_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        offsets,
        input_sizes,
        output_sizes,
        output_dim,
        batch_size);
  } else {
    remove_padding<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        offsets,
        input_sizes,
        output_sizes,
        output_dim,
        batch_size);
  }
}

template <typename T>
void remove_padding_transform0213_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  dim3 grid;
  grid.x = batch_size;
  grid.y = GRID_DIM_Y;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  TORCH_CHECK(
      output_dim == 2,
      "remove padding transform0213 only support output dim == 2");

  remove_padding_transform0213_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
      input,
      output,
      offsets,
      input_sizes,
      output_sizes,
      output_dim,
      batch_size);
}

template void remove_padding_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template void remove_padding_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template void remove_padding_transform0213_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template void remove_padding_transform0213_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template <typename T>
__global__ void add_padding_1(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int batch_output_offset = batch_id * output_sizes_1;
  for (int ii = 0; ii < (output_sizes_1 / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int output_offset = batch_output_offset + i;
    if (batch_id < batch_size && i < sizes_i[0]) {
      const int batch_input_offset = offsets[batch_id];
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
  const int i = (output_sizes_1 / grainsize) * grainsize + tid;
  if (i < output_sizes_1) {
    const int output_offset = batch_output_offset + i;
    if (batch_id < batch_size && (i < sizes_i[0])) {
      const int batch_input_offset = offsets[batch_id];
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
}

template <typename T>
__global__ void add_padding_2(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    int output_sizes_2,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int output_offset = batch_id * output_sizes_1 * output_sizes_2;
  const int output_numel = output_sizes_1 * output_sizes_2;
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes_2);
    const int i1 = i - i0 * output_sizes_2;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int offset = offsets[batch_id];
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes_2);
    const int i1 = i - i0 * output_sizes_2;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int offset = offsets[batch_id];
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template <typename T>
__global__ void add_padding_3(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    int output_sizes_2,
    int output_sizes_3,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int output_offset =
      batch_id * output_sizes_1 * output_sizes_2 * output_sizes_3;
  const int output_numel = output_sizes_1 * output_sizes_2 * output_sizes_3;
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes_2 * output_sizes_3);
    const int i1 = (i % (output_sizes_2 * output_sizes_3)) / output_sizes_3;
    const int i2 = i % output_sizes_3;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1] &&
        i2 < sizes_i[2]) {
      const int offset = offsets[batch_id];
      const int input_offset =
          offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes_2 * output_sizes_3);
    const int i1 = (i % (output_sizes_2 * output_sizes_3)) / output_sizes_3;
    const int i2 = i % output_sizes_3;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1] &&
        i2 < sizes_i[2]) {
      const int offset = offsets[batch_id];
      const int input_offset =
          offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template <typename T>
void add_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size) {
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  dim3 grid;
  grid.x = output_batch_size;
  grid.y = GRID_DIM_Y;
  if (input_dim == 1) {
    add_padding_1<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        batch_size);
  }
  if (input_dim == 2) {
    add_padding_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        output_sizes[2],
        batch_size);
  }
  if (input_dim == 3) {
    add_padding_3<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        output_sizes[2],
        output_sizes[3],
        batch_size);
  }
}

template void add_padding_kernelLauncher<double>(
    double* input,
    double* output,
    double padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

template void add_padding_kernelLauncher<float>(
    float* input,
    float* output,
    float padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

template void add_padding_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    c10::Half padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

// Passing lambda exp argument by value instead of by reference to avoid
// "internal compiler error: in maybe_undo_parenthesized_ref" error for specific
// compiler version.
#define JAGGED_TENSOR_DISPATCH_DIMS()                                         \
  AT_DISPATCH_INDEX_TYPES(x_offsets[0].scalar_type(), "jagged_indices", [=] { \
    switch (num_jagged_dim) {                                                 \
      case 1:                                                                 \
        INVOKE_KERNEL_WITH_DIM(1);                                            \
        break;                                                                \
      case 2:                                                                 \
        INVOKE_KERNEL_WITH_DIM(2);                                            \
        break;                                                                \
      case 3:                                                                 \
        INVOKE_KERNEL_WITH_DIM(3);                                            \
        break;                                                                \
      case 4:                                                                 \
        INVOKE_KERNEL_WITH_DIM(4);                                            \
        break;                                                                \
      case 5:                                                                 \
        INVOKE_KERNEL_WITH_DIM(5);                                            \
        break;                                                                \
      default:                                                                \
        TORCH_CHECK(                                                          \
            false, "unsupported number of jagged dim ", num_jagged_dim);      \
    }                                                                         \
  });

inline std::string torch_tensor_device_name(const at::Tensor& ten) {
  return c10::DeviceTypeName(ten.device().type());
}

inline std::string torch_tensor_device_name(
    const c10::optional<at::Tensor>& ten) {
  if (ten.has_value()) {
    return torch_tensor_device_name(ten.value());
  } else {
    return "N/A";
  }
}

inline bool torch_tensor_on_cuda_gpu_check(const at::Tensor& ten) {
  return ten.is_cuda();
}

inline bool torch_tensor_on_cuda_gpu_check(
    const c10::optional<at::Tensor>& ten) {
  return !ten.has_value() || torch_tensor_on_cuda_gpu_check(ten.value());
}

#define TENSOR_ON_CUDA_GPU(x)                                  \
  TORCH_CHECK(                                                 \
      torch_tensor_on_cuda_gpu_check(x),                       \
      #x " must be a CUDA tensor; it is currently on device ", \
      torch_tensor_device_name(x))

// A wrapper class for passing dynamically sized dimension information (e.g.
// tensor.dims()) from the host to device.
constexpr size_t kStackArrayMaxDims = 5;

template <typename T>
struct StackArray {
  T vals[kStackArrayMaxDims];
  size_t ndim;
};

// Warp size
#ifdef USE_ROCM
static constexpr int32_t kWarpSize = 64;
#else
static constexpr int32_t kWarpSize = 32;
#endif
// Max thread num in one thread block
static constexpr int32_t kMaxThreads = 1024;

#define DEVICE_INLINE __device__ inline __attribute__((always_inline))

__host__ DEVICE_INLINE int32_t div_round_up(int32_t a, int32_t b) {
  return (a + b - 1) / b;
}

inline std::tuple<dim3, dim3, StackArray<int64_t>> check_shape_and_partition_(
    const Tensor& values,
    const std::vector<Tensor>& offsets,
    const Tensor& dense_tensor) {
  const int outer_dense_size = dense_tensor.size(0);
  TORCH_CHECK(
      outer_dense_size == offsets[0].numel() - 1,
      "outer_dense_size, ",
      outer_dense_size,
      " != offsets[0].numel() - 1, ",
      offsets[0].numel() - 1);
  const int inner_dense_size = dense_tensor.size(-1);
  TORCH_CHECK(
      inner_dense_size == values.size(-1),
      "inner_dense_size, ",
      inner_dense_size,
      " != values.size(-1), ",
      values.size(-1));
  const int jagged_folded_size =
      dense_tensor.numel() / (outer_dense_size * inner_dense_size);

  const int threads_x =
      inner_dense_size >= kWarpSize / 2 ? kWarpSize : inner_dense_size;
  const int threads_y = kMaxThreads / kWarpSize;
  const dim3 blocks(
      div_round_up(outer_dense_size * jagged_folded_size, threads_y));

  StackArray<int64_t> jagged_dims_tensor;
  const int num_jagged_dim = dense_tensor.dim() - 2;
  TORCH_CHECK(num_jagged_dim <= kStackArrayMaxDims);
  jagged_dims_tensor.ndim = num_jagged_dim;
  std::memcpy(
      &(jagged_dims_tensor.vals[0]),
      dense_tensor.sizes().data() + 1,
      num_jagged_dim * sizeof(int64_t));
  return {dim3(threads_x, threads_y), blocks, jagged_dims_tensor};
}

template <int NUM_JAGGED_DIM, typename index_t>
DEVICE_INLINE bool walk_down_tensor_storage_tree_(
    int& offset,
    const int flattened_jagged_idx,
    const StackArray<int64_t>& jagged_dims,
    const StackArray<index_t*>& x_offsets) {
  // compute coorindates
  int jagged_coords[NUM_JAGGED_DIM];
  int j_temp = flattened_jagged_idx;
#pragma unroll
  for (int d = NUM_JAGGED_DIM - 1; d >= 0; --d) {
    const int jagged_size = jagged_dims.vals[d];
    jagged_coords[d] = j_temp % jagged_size;
    j_temp /= jagged_size;
  }

  // walk down the tree
  bool is_zero = false;
#pragma unroll
  for (int d = 0; d < NUM_JAGGED_DIM; ++d) {
    const int begin = x_offsets.vals[d][offset];
    const int end = x_offsets.vals[d][offset + 1];
    if (jagged_coords[d] >= end - begin) {
      is_zero = true;
      break;
    }
    offset = begin + jagged_coords[d];
  }
  return is_zero;
}

// output = f(x, y) where x is jagged, y is dense, and output is dense.
// A generic elementwise operation between a jagged tensor and a dense tensor
// This kernel assumes jagged dims are clustered together, preceded by outer
// dense dimensions and followed by inner dense dimensions.
// The outer/inner dense dimensions, and jagged dimensions in between are
// assumed to be folded so physically the dense tensor is 3D and the value of
// jagged tensor is 2D.
// To support arbitrary number of jagged dimensions, we pass a vector of
// pointers to offset tensors (this is ugly and probably we can use nested
// tensor here).
// This kernel parallelizes the (folded) inner dense dimension across
// blockDim.x so the inner dense dimension should be similar to or bigger than
// warp size.
// We rely on compiler unrolling the compiler time constant NUM_JAGGED_DIM.
template <int NUM_JAGGED_DIM, typename index_t, typename scalar_t, typename F>
__global__
__launch_bounds__(kMaxThreads) void jagged_dense_elementwise_dense_output_kernel_(
    const at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        x_values,
    StackArray<index_t*> x_offsets,
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> y,
    at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> output,
    StackArray<int64_t> jagged_dims,
    F f,
    const scalar_t padding_value) {
  const int outer_dense_size = y.size(0);
  const int jagged_folded_size = y.size(1);
  const int inner_dense_size = y.size(2);

  const int outer_begin = blockIdx.x * blockDim.y + threadIdx.y;
  const int outer_stride = gridDim.x * blockDim.y;
  for (int outer = outer_begin; outer < outer_dense_size * jagged_folded_size;
       outer += outer_stride) {
    const int oidx = outer / jagged_folded_size;
    const int jidx = outer % jagged_folded_size;

    int offset = oidx;
    const bool is_zero = walk_down_tensor_storage_tree_<NUM_JAGGED_DIM>(
        offset, jidx, jagged_dims, x_offsets);

    if (is_zero) {
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output[oidx][jidx][2 * iidx] =
            f(padding_value, y[oidx][jidx][2 * iidx]);
        output[oidx][jidx][2 * iidx + 1] =
            f(padding_value, y[oidx][jidx][2 * iidx + 1]);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output[oidx][jidx][2 * iidx] =
            f(padding_value, y[oidx][jidx][2 * iidx]);
      }
    } else {
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output[oidx][jidx][2 * iidx] =
            f(x_values[offset][2 * iidx], y[oidx][jidx][2 * iidx]);
        output[oidx][jidx][2 * iidx + 1] =
            f(x_values[offset][2 * iidx + 1], y[oidx][jidx][2 * iidx + 1]);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output[oidx][jidx][2 * iidx] =
            f(x_values[offset][2 * iidx], y[oidx][jidx][2 * iidx]);
      }
    }
  }
}

template <typename scalar_t, typename F>
void jagged_dense_elementwise_dense_output_(
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y,
    const Tensor& output,
    F f,
    const scalar_t padding_value = static_cast<scalar_t>(0)) {
  TENSOR_ON_CUDA_GPU(x_values);
  for (auto& x_offset : x_offsets) {
    TENSOR_ON_CUDA_GPU(x_offset);
  }

  const int num_jagged_dim = y.dim() - 2;
  TORCH_CHECK(
      x_offsets.size() == static_cast<size_t>(num_jagged_dim),
      "x_offsets.size(), ",
      x_offsets.size(),
      " != num_jagged_dim ",
      num_jagged_dim);

  if (y.numel() == 0) {
    return;
  }

  dim3 threads, blocks;
  StackArray<int64_t> jagged_dims_tensor;
  std::tie(threads, blocks, jagged_dims_tensor) =
      check_shape_and_partition_(x_values, x_offsets, y);

  // Canonicalize y and output to 3D, collapsing jagged dimensions.
  const Tensor y_reshaped = y.view({y.size(0), -1, y.size(-1)});
  Tensor output_reshaped = output.view(y_reshaped.sizes());

#define INVOKE_KERNEL_WITH_DIM(NUM_JAGGED_DIM)                                \
  {                                                                           \
    std::vector<Tensor> x_offsets_contig;                                     \
    x_offsets_contig.resize(num_jagged_dim);                                  \
    StackArray<index_t*> x_offset_ptrs;                                       \
    x_offset_ptrs.ndim = num_jagged_dim;                                      \
    for (int d = 0; d < num_jagged_dim; ++d) {                                \
      x_offsets_contig[d] = x_offsets[d].contiguous();                        \
      x_offset_ptrs.vals[d] =                                                 \
          x_offsets_contig[d].template data_ptr<index_t>();                   \
    }                                                                         \
    jagged_dense_elementwise_dense_output_kernel_<NUM_JAGGED_DIM, index_t>    \
        <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(           \
            x_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(), \
            x_offset_ptrs,                                                    \
            y_reshaped                                                        \
                .packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),     \
            output_reshaped                                                   \
                .packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),     \
            jagged_dims_tensor,                                               \
            f,                                                                \
            padding_value);                                                   \
  }

  JAGGED_TENSOR_DISPATCH_DIMS();
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#undef INVOKE_KERNEL_WITH_DIM
}

#define INVOKE_KERNEL_WITH_DIM(NUM_JAGGED_DIM)                                 \
  {                                                                            \
    dim3 threads, blocks;                                                      \
    StackArray<int64_t> jagged_dims_tensor;                                    \
    std::tie(threads, blocks, jagged_dims_tensor) =                            \
        check_shape_and_partition_(x_values, x_offsets, y);                    \
    blocks.x = div_round_up(x_values.size(0), threads.y);                      \
    std::vector<Tensor> x_offsets_contig;                                      \
    x_offsets_contig.resize(num_jagged_dim);                                   \
    StackArray<index_t*> x_offset_ptrs;                                        \
    x_offset_ptrs.ndim = num_jagged_dim;                                       \
    StackArray<int64_t> x_offset_sizes;                                        \
    x_offset_sizes.ndim = num_jagged_dim;                                      \
    for (int d = 0; d < num_jagged_dim; ++d) {                                 \
      x_offsets_contig[d] = x_offsets[d].contiguous();                         \
      x_offset_ptrs.vals[d] =                                                  \
          x_offsets_contig[d].template data_ptr<index_t>();                    \
      x_offset_sizes.vals[d] = x_offsets[d].numel();                           \
    }                                                                          \
    jagged_dense_dense_elementwise_jagged_output_kernel_<                      \
        NUM_JAGGED_DIM,                                                        \
        index_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(    \
        x_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),      \
        x_offset_ptrs,                                                         \
        x_offset_sizes,                                                        \
        y_reshaped.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),    \
        y_reshaped.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),    \
        output_values.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(), \
        jagged_dims_tensor,                                                    \
        [f] __device__(scalar_t x, scalar_t y, scalar_t /*unused*/)            \
            -> scalar_t { return f(x, y); });                                  \
  }

template <int NUM_JAGGED_DIM, typename index_t, typename scalar_t, typename F>
__global__
__launch_bounds__(kMaxThreads) void jagged_dense_dense_elementwise_jagged_output_kernel_(
    const at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        x_values,
    StackArray<index_t*> x_offsets,
    StackArray<int64_t> x_offsets_sizes,
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> y_0,
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits> y_1,
    at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        output_values,
    StackArray<int64_t> jagged_dims,
    F f) {
  const int outer_dense_size = y_0.size(0);
  const int inner_dense_size = y_0.size(2);
  const int nnz = x_values.size(0);

  const int offset_begin = blockIdx.x * blockDim.y + threadIdx.y;
  const int offset_stride = gridDim.x * blockDim.y;
  for (int offset = offset_begin; offset < nnz; offset += offset_stride) {
    int offset_temp = offset;
    int jidx = 0;
    bool truncated = false;
    int dim_prod = 1;
#pragma unroll
    for (int d = NUM_JAGGED_DIM - 1; d >= 0; --d) {
      // Binary search the first that is bigger than offset
      int count = x_offsets_sizes.vals[d] - 1;
      int first = 1;
      while (count > 0) {
        int idx = first;
        int step = count / 2;
        idx += step;
        if (x_offsets.vals[d][idx] <= offset_temp) {
          first = ++idx;
          count -= step + 1;
        } else {
          count = step;
        }
      }

      --first;
      int coord = offset_temp - x_offsets.vals[d][first];
      if (coord >= jagged_dims.vals[d]) {
        truncated = true;
        break;
      }
      jidx += coord * dim_prod;
      dim_prod *= jagged_dims.vals[d];
      offset_temp = first;
    }

    if (offset_temp >= outer_dense_size) {
      // This can happen when values have more elements than the last element of
      // offset
      truncated = true;
    }
    if (!truncated) {
      const int oidx = offset_temp;
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output_values[offset][2 * iidx] =
            f(x_values[offset][2 * iidx],
              y_0[oidx][jidx][2 * iidx],
              y_1[oidx][jidx][2 * iidx]);
        output_values[offset][2 * iidx + 1] =
            f(x_values[offset][2 * iidx + 1],
              y_0[oidx][jidx][2 * iidx + 1],
              y_1[oidx][jidx][2 * iidx + 1]);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output_values[offset][2 * iidx] =
            f(x_values[offset][2 * iidx],
              y_0[oidx][jidx][2 * iidx],
              y_1[oidx][jidx][2 * iidx]);
      }
    } else {
      int iidx;
      for (iidx = threadIdx.x; iidx * 2 + 1 < inner_dense_size;
           iidx += blockDim.x) {
        output_values[offset][2 * iidx] = f(x_values[offset][2 * iidx], 0, 0);
        output_values[offset][2 * iidx + 1] =
            f(x_values[offset][2 * iidx + 1], 0, 0);
      }
      if (iidx * 2 + 1 == inner_dense_size) {
        output_values[offset][2 * iidx] = f(x_values[offset][2 * iidx], 0, 0);
      }
    }
  }
}

///@addtogroup jagged-tensor-ops-cuda
template <typename scalar_t, typename F>
void jagged_dense_elementwise_jagged_output_(
    const Tensor& x_values,
    const std::vector<Tensor>& x_offsets,
    const Tensor& y,
    const Tensor& output_values,
    F f) {
  TENSOR_ON_CUDA_GPU(x_values);
  for (auto& x_offset : x_offsets) {
    TENSOR_ON_CUDA_GPU(x_offset);
  }

  const int num_jagged_dim = y.dim() - 2;
  TORCH_CHECK(
      x_offsets.size() == static_cast<size_t>(num_jagged_dim),
      "x_offsets.size(), ",
      x_offsets.size(),
      " != num_jagged_dim, ",
      num_jagged_dim);

  if (y.numel() == 0 || x_values.numel() == 0) {
    return;
  }

  // Canonicalize y to 3D, collapsing jagged dimensions.
  const Tensor y_reshaped = y.view({y.size(0), -1, y.size(-1)});

  JAGGED_TENSOR_DISPATCH_DIMS();
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

#undef INVOKE_KERNEL_WITH_DIM

at::Tensor _fbgemm_jagged_to_padded_dense_forward(
    const Tensor& values,
    TensorList offsets,
    c10::IntArrayRef max_lengths,
    const double padding_value) {
  const size_t num_jagged_dim = offsets.size();
  TORCH_CHECK(
      max_lengths.size() == num_jagged_dim,
      "max_lengths.size(), ",
      max_lengths.size(),
      " != num_jagged_dim, ",
      num_jagged_dim);
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(values.get_device());

  const Tensor values_canonicalized = values.view(
      {values.size(0),
       std::accumulate(
           values.sizes().begin() + 1,
           values.sizes().end(),
           1,
           std::multiplies<size_t>())});
  at::SymDimVector padded_values_shape({at::SymInt(offsets[0].size(0) - 1)});
  padded_values_shape.insert(
      padded_values_shape.end(), max_lengths.begin(), max_lengths.end());

  // Canonicalize padded_values by unsqueeze the last dim if the inner dense
  // dimension is 1 and folded.
  const bool D_folded = values.dim() == 1;
  if (!D_folded) {
    padded_values_shape.push_back(values.size(-1));
  }
  Tensor padded_values =
      at::empty_symint(padded_values_shape, values.options());
  Tensor padded_values_view =
      D_folded ? padded_values.unsqueeze(-1) : padded_values;

  AT_DISPATCH_ALL_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      values.scalar_type(),
      "jagged_to_padded_dense",
      [&] {
        jagged_dense_elementwise_dense_output_<scalar_t>(
            values_canonicalized,
            offsets.vec(),
            padded_values_view, // dummy not used in the lambda function
            padded_values_view,
           [] __device__(scalar_t x, scalar_t /*unused*/) -> scalar_t {
              return x;
            },
            static_cast<scalar_t>(padding_value));
      });

  return padded_values;
}

at::Tensor _fbgemm_jagged_to_padded_dense_backward(
    const Tensor& grad_output,
    TensorList offsets,
    int64_t total_L) {
  auto grad_padded_values = grad_output;
  at::cuda::OptionalCUDAGuard device_guard;
  device_guard.set_index(grad_padded_values.get_device());

  // Canonicalize padded_values by unsqueeze the last dim if the inner dense
  // dimension is 1 and folded.
  const bool D_folded = grad_padded_values.dim() == offsets.size() + 1;
  Tensor grad_padded_values_view =
      D_folded ? grad_padded_values.unsqueeze(-1) : grad_padded_values;
  int32_t D = grad_padded_values_view.size(-1);

  // Initialize with zeros so output will be zero for the portion truncated
  // in forward.
  auto grad_values =
      at::zeros_symint({total_L, D}, grad_padded_values.options());

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      grad_padded_values.scalar_type(),
      "jagged_to_dense_backward_kernel",
      [&] {
        jagged_dense_elementwise_jagged_output_<scalar_t>(
            grad_values, // dummy not used in the lambda function
            offsets.vec(),
            grad_padded_values_view,
            grad_values,
            [] __device__(scalar_t /*unused*/, scalar_t y) -> scalar_t {
              return y;
            });
      });

  return D_folded ? grad_values.squeeze(-1) : grad_values;
}

} // namespace native
} // namespace at
